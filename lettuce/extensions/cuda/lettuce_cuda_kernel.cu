#include "hip/hip_runtime.h"
#if _MSC_VER && !__INTEL_COMPILER
#pragma warning ( push )
#pragma warning ( disable : 4067 )
#pragma warning ( disable : 4624 )
#endif

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if _MSC_VER && !__INTEL_COMPILER
#pragma warning ( pop )
#endif

#include "lettuce_cuda_stencil.cu"
#include "lettuce_cuda_lattice.cu"

using index_t = unsigned int;

template<typename scalar_t, index_t D, index_t Q, stencil<scalar_t, D, Q> STENCIL>
__device__ __forceinline__ void
quadratic_equilibrium_collision(scalar_t *f, scalar_t tau)
{
    /*
     * avoid recalculating the inverse of tau
     */

    const scalar_t tau_inv = 1.0 / tau;

    /*
     * begin calculating the equilibrium
     */

    constexpr auto cs_pow_two = STENCIL.cs * STENCIL.cs;
    constexpr auto two_cs_pow_two = cs_pow_two + cs_pow_two;

    scalar_t rho;
    lattice::rho<scalar_t, D>(f, &rho);

    scalar_t j[2];
    lattice::j<scalar_t, D, Q, STENCIL>(f, &j);

    const scalar_t u[2] = {j[0] / rho, j[1] / rho};
    const scalar_t uxu = u[0] * u[0] + u[1] * u[1];

#pragma unroll
    for (index_t i = 0; i < 9; ++i)
    {
        const scalar_t exu = STENCIL.e[i][0] * u[0] + STENCIL.e[i][1] * u[1];

        const scalar_t tmp0 = exu / cs_pow_two;
        const scalar_t tmp1 = rho * (((exu + exu - uxu) / two_cs_pow_two) + (0.5 * (tmp0 * tmp0)) + 1.0);
        const scalar_t feq = tmp1 * STENCIL.w[i];

        /*
         * finally apply the collision operator
         */

        f[i] = f[i] - (tau_inv * (f[i] - feq));
    }
}

template<typename scalar_t>
__device__ __forceinline__ void
d2q9_read(const scalar_t *f, scalar_t *f_reg, index_t length, index_t index)
{
    // the reading index is trivial as it is the same relative index in each dimension
    // [by using an iterator bypass some multiplications]
    auto index_it = index; f_reg[0] = f[index_it];
    index_it += length;    f_reg[1] = f[index_it];
    index_it += length;    f_reg[2] = f[index_it];
    index_it += length;    f_reg[3] = f[index_it];
    index_it += length;    f_reg[4] = f[index_it];
    index_it += length;    f_reg[5] = f[index_it];
    index_it += length;    f_reg[6] = f[index_it];
    index_it += length;    f_reg[7] = f[index_it];
    index_it += length;    f_reg[8] = f[index_it];
}

template<typename scalar_t>
__device__ __forceinline__ void
d2q9_standard_stream_read(
        const scalar_t *f, scalar_t *f_reg,
        index_t width, index_t height, index_t length,
        index_t index, index_t horizontal_index, index_t vertical_index,
        index_t vertical_m_offset)
{
    /*
     * define needed variables for the streaming
     */

    // alter name for convenience
    const auto &horizontal_m_offset = horizontal_index;

    // pre calculate the vertical and horizontal offsets
    const auto vertical_t_offset = ((vertical_index == 0) ? height - 1 : (vertical_index - 1)) * width;
    const auto vertical_b_offset = (((vertical_index + 1) == height) ? 0 : (vertical_index + 1)) * width;
    const auto horizontal_l_offset = (horizontal_index == 0) ? width - 1 : (horizontal_index - 1);
    const auto horizontal_r_offset = ((horizontal_index + 1) == width) ? 0 : (horizontal_index + 1);

    /*
     * read the neighbor distributions into the current/register node
     */

    // center force is trivial as it stays in place
    f_reg[0] = f[index];

    // the index from which to stream from is calculated by:
    // - a dimensional offset (which is calculated by iteration)
    //   [by using an iterator bypass some multiplications]
    // - a relative horizontal offset (corresponding to the dimension)
    // - a relative vertical offset (corresponding to the dimension)
    auto dim_offset_it = length;
    f_reg[1] = f[dim_offset_it + horizontal_m_offset + vertical_t_offset];
    dim_offset_it += length;
    f_reg[2] = f[dim_offset_it + horizontal_l_offset + vertical_m_offset];
    dim_offset_it += length;
    f_reg[3] = f[dim_offset_it + horizontal_m_offset + vertical_b_offset];
    dim_offset_it += length;
    f_reg[4] = f[dim_offset_it + horizontal_r_offset + vertical_m_offset];
    dim_offset_it += length;
    f_reg[5] = f[dim_offset_it + horizontal_l_offset + vertical_t_offset];
    dim_offset_it += length;
    f_reg[6] = f[dim_offset_it + horizontal_l_offset + vertical_b_offset];
    dim_offset_it += length;
    f_reg[7] = f[dim_offset_it + horizontal_r_offset + vertical_b_offset];
    dim_offset_it += length;
    f_reg[8] = f[dim_offset_it + horizontal_r_offset + vertical_t_offset];
}

template<typename scalar_t>
__device__ __forceinline__ void
d2q9_write(const scalar_t *f_reg, scalar_t *f_next, index_t length, index_t index)
{
    // the writing index is trivial as it is the same relative index in each dimension
    // [by using an iterator bypass some multiplications]
    auto index_it = index;
    f_next[index_it] = f_reg[0];
    index_it += length;
    f_next[index_it] = f_reg[1];
    index_it += length;
    f_next[index_it] = f_reg[2];
    index_it += length;
    f_next[index_it] = f_reg[3];
    index_it += length;
    f_next[index_it] = f_reg[4];
    index_it += length;
    f_next[index_it] = f_reg[5];
    index_it += length;
    f_next[index_it] = f_reg[6];
    index_it += length;
    f_next[index_it] = f_reg[7];
    index_it += length;
    f_next[index_it] = f_reg[8];
}

/**
 * collide and stream the given field (f)
 *
 * steps: TODO out of date documentation
 * 1. read all nodes (from f)
 * 2. add collision value (local)
 * 3. stream values (local)
 * 4. write all nodes (to f_next)
 *
 * @tparam scalar_t the scalar type of the tensor passed. typically defined by AT_DISPATCH_FLOATING_TYPES
 * @param f the fluid forces at time t (at the moment)
 * @param f_next a memory region as big as f which is used to write the simulation results into
 * @param tau TODO document better what tau is
 * @param width the width of the field
 * @param length the length of the memory region (f/f_next) (second dimension) which is equal to with*height of the field
 */
template<typename scalar_t>
__global__ void
lettuce_cuda_stream_and_collide_kernel(const scalar_t *f, scalar_t *f_next, scalar_t tau, index_t width, index_t height, index_t length)
{
    /*
     * define needed variables for the streaming
     */

    // pre calculate the vertical and horizontal indices before streaming
    const auto horizontal_index = blockIdx.x * blockDim.x + threadIdx.x;
    const auto vertical_index = blockIdx.y * blockDim.y + threadIdx.y;

    // pre calculate the vertical and horizontal offsets before streaming
    const auto &horizontal_m_offset = horizontal_index;
    const auto vertical_m_offset = vertical_index * width;

    // pre calculate the current index
    const auto index = vertical_m_offset + horizontal_m_offset;

    /*
     * do the work
     */

    // standard stream & read
    scalar_t f_reg[9];
    d2q9_standard_stream_read(f, &(f_reg[0]), width, height, length, index, horizontal_index, vertical_index, vertical_m_offset);

    // collide & write
    quadratic_equilibrium_collision<scalar_t, 2, 9, d2q9<scalar_t>>(&(f_reg[0]), tau);
    d2q9_write(f_reg, f_next, length, index);
}

/**
 * TODO document
 */
template<typename scalar_t>
__global__ void
lettuce_cuda_stream_kernel(const scalar_t *f, scalar_t *f_next, index_t width, index_t height, index_t length)
{
    /*
     * define needed variables for the streaming
     */

    // pre calculate the vertical and horizontal indices before streaming
    const auto horizontal_index = blockIdx.x * blockDim.x + threadIdx.x;
    const auto vertical_index = blockIdx.y * blockDim.y + threadIdx.y;

    // pre calculate the vertical and horizontal offsets before streaming
    const auto &horizontal_m_offset = horizontal_index;
    const auto vertical_m_offset = vertical_index * width;

    // pre calculate the current index
    const auto index = vertical_m_offset + horizontal_m_offset;

    /*
     * do the work
     */

    // standard stream & read & write
    scalar_t f_reg[9];
    d2q9_standard_stream_read(f, &(f_reg[0]), width, height, length, index, horizontal_index, vertical_index, vertical_m_offset);
    d2q9_write(f_reg, f_next, length, index);
}

/**
 * TODO document
 */
template<typename scalar_t>
__global__ void
lettuce_cuda_collide_kernel(const scalar_t *f, scalar_t *f_next, scalar_t tau, index_t width, index_t height, index_t length)
{
    /*
     * define needed variables for the streaming
     */

    // pre calculate the vertical and horizontal indices before streaming
    const auto horizontal_index = blockIdx.x * blockDim.x + threadIdx.x;
    const auto vertical_index = blockIdx.y * blockDim.y + threadIdx.y;

    // pre calculate the vertical and horizontal offsets before streaming
    const auto &horizontal_m_offset = horizontal_index;
    const auto vertical_m_offset = vertical_index * width;

    // pre calculate the current index
    const auto index = vertical_m_offset + horizontal_m_offset;

    /*
     * do the work
     */

    // read
    scalar_t f_reg[9];
    d2q9_read(f, &(f_reg[0]), length, index);

    // collide & write
    quadratic_equilibrium_collision<scalar_t, 2, 9, d2q9<scalar_t>>(&(f_reg[0]), tau);
    d2q9_write(f_reg, f_next, length, index);
}

void
lettuce_cuda_stream_and_collide(at::Tensor f, at::Tensor f_next, double tau)
{
    /*
     * Use all threads of one block (asserting the block support 1024 threads)
     */

    const auto thread_count = dim3{16u, 16u};

    /*
     * calculate constant values
     */

    const auto width = static_cast<index_t> (f.sizes()[1]);
    const auto height = static_cast<index_t> (f.sizes()[2]);

    const auto block_count = ([&]()
    {
        assert((width % thread_count.x) == 0u);
        assert((height % thread_count.y) == 0u);

        const auto horizontal_block_count = width / thread_count.x;
        const auto vertical_block_count = height / thread_count.y;

        return dim3{horizontal_block_count, vertical_block_count};
    }());

    /*
     * call the cuda kernel in a safe way for all supported float types
     */

    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "lettuce_cuda_stream_and_collide", ([&]
    {
        lettuce_cuda_stream_and_collide_kernel<scalar_t><<<block_count, thread_count>>>(
                f.data<scalar_t>(),
                f_next.data<scalar_t>(),
                static_cast<scalar_t>(tau),
                width,
                height,
                width * height
        );
        hipDeviceSynchronize(); // TODO maybe replace with torch.cuda.sync().
        //      this may be more efficient as is bridges
        //      the time to return from the native code.
        //      but maybe this time is not noticeable ...
    }));
}

void
lettuce_cuda_stream(at::Tensor f, at::Tensor f_next)
{
    /*
     * Use all threads of one block (asserting the block support 1024 threads)
     */

    const auto thread_count = dim3{16u, 16u};

    /*
     * calculate constant values
     */

    const auto width = static_cast<index_t> (f.sizes()[1]);
    const auto height = static_cast<index_t> (f.sizes()[2]);

    const auto block_count = ([&]()
    {
        assert((width % thread_count.x) == 0u);
        assert((height % thread_count.y) == 0u);

        const auto horizontal_block_count = width / thread_count.x;
        const auto vertical_block_count = height / thread_count.y;

        return dim3{horizontal_block_count, vertical_block_count};
    }());

    /*
     * call the cuda kernel in a safe way for all supported float types
     */

    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "lettuce_cuda_stream_and_collide", ([&]
    {
        lettuce_cuda_stream_kernel<scalar_t><<<block_count, thread_count>>>(
                f.data<scalar_t>(),
                f_next.data<scalar_t>(),
                width,
                height,
                width * height
        );
        hipDeviceSynchronize(); // TODO maybe replace with torch.cuda.sync().
        //      this may be more efficient as is bridges
        //      the time to return from the native code.
        //      but maybe this time is not noticeable ...
    }));
}

void
lettuce_cuda_collide(at::Tensor f, at::Tensor f_next, double tau)
{
    /*
     * Use all threads of one block (asserting the block support 1024 threads)
     */

    const auto thread_count = dim3{16u, 16u};

    /*
     * calculate constant values
     */

    const auto width = static_cast<index_t> (f.sizes()[1]);
    const auto height = static_cast<index_t> (f.sizes()[2]);

    const auto block_count = ([&]()
    {
        assert((width % thread_count.x) == 0u);
        assert((height % thread_count.y) == 0u);

        const auto horizontal_block_count = width / thread_count.x;
        const auto vertical_block_count = height / thread_count.y;

        return dim3{horizontal_block_count, vertical_block_count};
    }());

    /*
     * call the cuda kernel in a safe way for all supported float types
     */

    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "lettuce_cuda_stream_and_collide", ([&]
    {
        lettuce_cuda_collide_kernel<scalar_t><<<block_count, thread_count>>>(
                f.data<scalar_t>(),
                f_next.data<scalar_t>(),
                static_cast<scalar_t>(tau),
                width,
                height,
                width * height
        );
        hipDeviceSynchronize(); // TODO maybe replace with torch.cuda.sync().
        //      this may be more efficient as is bridges
        //      the time to return from the native code.
        //      but maybe this time is not noticeable ...
    }));
}
